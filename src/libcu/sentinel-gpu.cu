#include "hip/hip_runtime.h"
#include <crtdefscu.h>
#include <stdiocu.h>
#include <stdlibcu.h>
#include <sentinel.h>

#define DEVICE_SPINLOCK(SET, WHEN) do { s_ = *control; /*printf("%d ", s_);*/ __syncthreads(); } while (s_ != WHEN); *control = SET; __syncthreads();

__BEGIN_DECLS;

#if HAS_DEVICESENTINEL

static __device__ char *executeTrans(sentinelCommand *cmd, int size, sentinelInPtr *listIn, sentinelOutPtr *listOut, intptr_t offset);
static __device__ char *preparePtrs(sentinelInPtr *ptrsIn, sentinelOutPtr *ptrsOut, sentinelCommand *cmd, char *data, char *dataEnd, intptr_t offset, sentinelOutPtr *&transListOut) {
	char *ptr = data, *next;
	int transSize = 0;
	sentinelInPtr *listIn = nullptr;
	sentinelOutPtr *listOut = nullptr;

	// PREPARE
	if (ptrsIn)
		for (sentinelInPtr *p = ptrsIn; p->field; p++) {
			char **field = (char **)p->field;
			int size = p->size != -1 ? p->size : (p->size = *field ? (int)strlen(*field) + 1 : 0);
			next = ptr + size;
			if (!size)
				p->unknown = nullptr;
			else if (next <= dataEnd) {
				p->unknown = ptr;
				ptr = next;
			}
			else {
				p->unknown = listIn; listIn = p;
				transSize += size;
			}
		}
	if (ptrsOut) {
		ptr = ptrsOut[0].field == (char *)-1 ? ptr : data;
		for (sentinelOutPtr *p = ptrsOut; p->field; p++) {
			char **field = (char **)p->field;
			int size = p->size != -1 ? p->size : dataEnd - ptr;
			next = ptr + size;
			if (!size) {}
			else if (next <= dataEnd) {
				*field = ptr + offset;
				ptr = next;
			}
			else {
				p->unknown = listOut; listOut = p;
				transSize += size;
			}
		}
		transListOut = listOut;
	}

	// TRANSFER IN
	if (transSize)
		executeTrans(cmd, transSize, listIn, nullptr, offset);

	// PACK
	for (sentinelInPtr *p = ptrsIn; p->field; p++) {
		char **field = (char **)p->field;
		char *ptr = (char *)p->unknown;
		if (!ptr || !*field)
			continue;
		memcpy(ptr, *field, p->size);
		*field = ptr + offset;
	}
	return data;
}

static __device__ bool postfixPtrs(sentinelOutPtr *ptrsOut, sentinelCommand *cmd, intptr_t offset) {
	// UNPACK
	for (sentinelOutPtr *p = ptrsOut; p->field; p++) {
		char **buf = (char **)p->buf;
		if (!*buf)
			continue;
		char **field = (char **)p->field;
		char *ptr = *field - offset;
		int *sizeField = (int *)p->sizeField;
		int size = !*sizeField ? p->size : *sizeField;
		if (size > 0) memcpy(*buf, ptr, size);
	}
	return true;
}

__device__ volatile unsigned int _sentinelMapId;
__constant__ const sentinelMap *_sentinelDeviceMap[SENTINEL_DEVICEMAPS];
__device__ void sentinelDeviceSend(sentinelMessage *msg, int msgLength, sentinelInPtr *ptrsIn, sentinelOutPtr *ptrsOut) {
	unsigned int s_;
	const sentinelMap *map = _sentinelDeviceMap[_sentinelMapId++ % SENTINEL_DEVICEMAPS];
	if (!map)
		panic("sentinel: device map not defined. did you start sentinel?\n");

	// ATTACH
	long id = atomicAdd((int *)&map->setId, SENTINEL_MSGSIZE);
	sentinelCommand *cmd = (sentinelCommand *)&map->data[id % sizeof(map->data)];
	if (cmd->magic != SENTINEL_MAGIC)
		panic("bad sentinel magic");
	int *unknown = &cmd->unknown; volatile long *control = (volatile long *)&cmd->control; intptr_t offset = map->offset;
	DEVICE_SPINLOCK(SENTINELCONTROL_DEVICE, SENTINELCONTROL_NORMAL);

	// PREPARE
	cmd->length = msgLength;
	char *data = cmd->data + ROUND8_(msgLength), *dataEnd = data + msg->size;
	sentinelOutPtr *transListOut = nullptr;
	if (((ptrsIn || ptrsOut) && !(data = preparePtrs(ptrsIn, ptrsOut, cmd, data, dataEnd, offset, transListOut))) ||
		(msg->prepare && !msg->prepare(msg, data, dataEnd, offset)))
		panic("msg too long");
	memcpy(cmd->data, msg, msgLength);
	//printf("msg: %d[%d]'", msg->op, msgLength); for (int i = 0; i < msgLength; i++) printf("%02x", ((char *)msg)[i] & 0xff); printf("'\n");
	*unknown = 0; *control = SENTINELCONTROL_DEVICERDY;

	// FLOW-WAIT
	if (msg->flow & SENTINELFLOW_WAIT) {
		DEVICE_SPINLOCK(SENTINELCONTROL_DEVICE, SENTINELCONTROL_HOSTRDY);
		executeTrans(cmd, 0, nullptr, transListOut, offset);
		memcpy(msg, cmd->data, msgLength);
		if ((ptrsOut && !postfixPtrs(ptrsOut, cmd, offset)) ||
			(msg->postfix && !msg->postfix(msg, offset)))
			panic("postfix error");
		*unknown = 0; *control = SENTINELCONTROL_DEVICERDY;
	}
	*control = SENTINELCONTROL_NORMAL;
}

static __device__ char *executeTrans(sentinelCommand *cmd, int size, sentinelInPtr *listIn, sentinelOutPtr *listOut, intptr_t offset) {
	unsigned int s_;
	int *unknown = &cmd->unknown; volatile long *control = (volatile long *)&cmd->control;
	char *data = cmd->data;
	// create memory
	if (size) {
		*(int *)data = size;
		*unknown = 1; *control = SENTINELCONTROL_DEVICERDY;
		DEVICE_SPINLOCK(SENTINELCONTROL_DEVICE, SENTINELCONTROL_HOSTRDY);
	}
	char *ptr = *(char **)data;
	// transfer
	if (listIn) {
		for (sentinelInPtr *p = listIn; p; p = (sentinelInPtr *)p->unknown) {
			char **field = (char **)p->field;
			int size = p->size, length = 0; const char *v = (const char *)*field;
			while (size > 0) {
				length = cmd->length = size > SENTINEL_MSGSIZE ? SENTINEL_MSGSIZE : size;
				memcpy(data, (void *)v, length); size -= length; v += length;
				*unknown = 2; *control = SENTINELCONTROL_DEVICERDY;
				DEVICE_SPINLOCK(SENTINELCONTROL_DEVICE, SENTINELCONTROL_HOSTRDY);
			}
			*field = ptr; ptr += size;
			p->unknown = nullptr;
		}
		*unknown = 0; *control = SENTINELCONTROL_DEVICERDY;
	}
	if (listOut) {
		for (sentinelOutPtr *p = listOut; p; p = (sentinelOutPtr *)p->unknown) {
			char **field = (char **)p->field;
			int size = p->size, length = 0; const char *v = (const char *)*field;
			while (size > 0) {
				length = cmd->length = size > SENTINEL_MSGSIZE ? SENTINEL_MSGSIZE : size;
				memcpy((void *)v, data, length); size -= length; v += length;
				*unknown = 3; *control = SENTINELCONTROL_DEVICERDY;
				DEVICE_SPINLOCK(SENTINELCONTROL_DEVICE, SENTINELCONTROL_HOSTRDY);
			}
			*field = ptr; ptr += size;
			p->unknown = nullptr;
		}
	}
}

#endif

__END_DECLS;
