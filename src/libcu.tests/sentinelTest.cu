#include "hip/hip_runtime.h"
#include <stdiocu.h>
#include <stringcu.h>
#include <sentinel.h>
#include <assert.h>

enum {
	MODULE_SIMPLE = 500,
	MODULE_STRING,
	MODULE_RETURN,
	MODULE_CUSTOM,
	MODULE_COMPLEX,
	MODULE_JUMBOOUT,
};

struct module_simple {
	sentinelMessage base;
	int value;
	__device__ module_simple(bool wait, int value) : base(MODULE_SIMPLE, wait ? SENTINELFLOW_WAIT : SENTINELFLOW_NONE), value(value) { sentinelDeviceSend(&base, sizeof(module_simple)); }
	int rc;
};

struct module_string {
	sentinelMessage base;
	const char *str;
	__device__ module_string(bool wait, const char *str) : base(MODULE_STRING, wait ? SENTINELFLOW_WAIT : SENTINELFLOW_NONE, SENTINEL_CHUNK), str(str) { sentinelDeviceSend(&base, sizeof(module_string), ptrsIn); }
	int rc;
	sentinelInPtr ptrsIn[2] = {
		{ &str, -1 },
		{ nullptr }
	};
};

struct module_return {
	sentinelMessage base;
	const char *buf; size_t size;
	__device__ module_return(const char *buf, size_t size) : base(MODULE_RETURN, SENTINELFLOW_WAIT, SENTINEL_CHUNK), buf(buf), size(size) { ptrsOut[0].size = size; sentinelDeviceSend(&base, sizeof(module_return), nullptr, ptrsOut); }
	size_t rc;
	void *ptr;
	sentinelOutPtr ptrsOut[2] = {
		{ &ptr, &buf, 0 },
		{ nullptr }
	};
};

struct module_custom {
	static __forceinline__ __device__ char *prepare(module_custom *t, char *data, char *dataEnd, intptr_t offset) {
		int strLength = t->str ? (int)strlen(t->str) + 1 : 0;
		char *str = data;
		char *end = data += strLength;
		if (end > dataEnd) return nullptr;
		memcpy(str, t->str, strLength);
		t->str = str + offset;
		t->ptr = str + offset;
		return end;
	}
	static __forceinline__ __device__ bool postfix(module_custom *t, intptr_t offset) {
		char *ptr = (char *)t->ptr - offset;
		//if (t->rc > 0) memcpy(t->buf, ptr, t->rc);
		return true;
	}
	sentinelMessage base;
	const char *str; char *buf;
	__device__ module_custom(bool wait, const char *str) : base(MODULE_CUSTOM, wait ? SENTINELFLOW_WAIT : SENTINELFLOW_NONE, SENTINEL_CHUNK, SENTINELPREPARE(prepare), SENTINELPOSTFIX(postfix)), str(str), buf(buf) { sentinelDeviceSend(&base, sizeof(module_custom)); }
	int rc;
	void *ptr;
};

struct module_complex {
	sentinelMessage base;
	const char *str;
	__device__ module_complex(bool wait, const char *str, char *ptr, size_t size) : base(MODULE_COMPLEX, wait ? SENTINELFLOW_WAIT : SENTINELFLOW_NONE, SENTINEL_CHUNK), str(str), ptr(ptr) { ptrsIn[1].size = size; sentinelDeviceSend(&base, sizeof(module_complex), ptrsIn); }
	int rc; int rc2;
	void *ptr;
	sentinelInPtr ptrsIn[3] = {
		{ &str, -1 },
		{ &ptr, 0 },
		{ nullptr }
	};
};

bool sentinelModuleExecutor(void *tag, sentinelMessage *data, int length, char *(**hostPrepare)(void*, char*, char*, intptr_t)) {
	switch (data->op) {
	case MODULE_SIMPLE: { module_simple *msg = (module_simple *)data; msg->rc = msg->value; return true; }
	case MODULE_STRING: { module_string *msg = (module_string *)data; msg->rc = (int)strlen(msg->str); return true; }
	case MODULE_RETURN: { module_return *msg = (module_return *)data; msg->rc = 5; strcpy((char *)msg->ptr, "test"); return true; }
	case MODULE_CUSTOM: { module_custom *msg = (module_custom *)data; msg->rc = (int)strlen(msg->str); return true; }
	case MODULE_COMPLEX: { module_complex *msg = (module_complex *)data; msg->rc = (int)strlen(msg->str); msg->rc2 = (int)strlen((char *)msg->ptr); return true; }
	}
	return false;
}
static sentinelExecutor _moduleExecutor = { nullptr, "module", sentinelModuleExecutor, nullptr };

static __global__ void g_sentinel_test1() {
	//printf("sentinel_test1\n");

	//// SENTINELDEVICESEND ////
	//	extern __device__ void sentinelDeviceSend(sentinelMessage *msg, int msgLength);
	for (int k = 0; k < 0; k++) {
		printf("%d ", k);
		char buf[100];
		module_simple a0(true, 1); int a0a = a0.rc; assert(a0a == 1);
		module_string a1(true, "test"); int a1a = a1.rc; assert(a1a == 4);
		module_return a2(buf, sizeof(buf)); int a2a = a2.rc; assert(a2a == 5 && !strcmp(buf, "test"));
		module_custom a3(true, "test"); int a3a = a3.rc; assert(a3a == 4);
		char complex[2048]; memset(complex, 1, sizeof(complex));
		module_complex a4(true, "test", complex, sizeof(complex)); int a4a = a4.rc; assert(a4a == 4);
	}

	// JUMBO
	for (int k = 0; k < 100; k++) {
		printf("J%d ", k);
		char jumbo[9046]; memset(jumbo, 2, sizeof(jumbo)); jumbo[9045] = 0;
		//module_string a1(true, "test"); int a1a = a1.rc; assert(a1a == 4);
		module_complex b0(true, "test", jumbo, sizeof(jumbo)); int b0a = b0.rc; int b0b = b0.rc2; assert(b0a == 4 && b0b == 9045);
		//module_return b1(jumbo, sizeof(jumbo)); int b1a = b1.rc; assert(b1a == 5 && !strcmp(jumbo, "test"));
	}
}

hipError_t sentinel_test1() {
	sentinelRegisterExecutor(&_moduleExecutor);
	g_sentinel_test1<<<1, 1>>>(); return hipDeviceSynchronize();
}

//// SENTINELDEFAULTEXECUTOR ////
//	extern bool sentinelDefaultExecutor(void *tag, sentinelMessage *data, int length, char *(**hostPrepare)(void*,char*,char*,intptr_t));

//// SENTINELSERVERINITIALIZE, SENTINELSERVERSHUTDOWN ////
//	extern void sentinelServerInitialize(sentinelExecutor *executor = nullptr, char *mapHostName = SENTINEL_NAME, bool hostSentinel = true, bool deviceSentinel = true);
//	extern void sentinelServerShutdown();

//// SENTINELDEVICESEND ////
//	extern __device__ void sentinelDeviceSend(sentinelMessage *msg, int msgLength);

//// SENTINELCLIENTINITIALIZE, SENTINELCLIENTSHUTDOWN ////
//	extern void sentinelClientInitialize(char *mapHostName = SENTINEL_NAME);
//	extern void sentinelClientShutdown();

//// SENTINELCLIENTSEND ////
//	extern void sentinelClientSend(sentinelMessage *msg, int msgLength);

//// SENTINELFINDEXECUTOR, SENTINELREGISTEREXECUTOR, SENTINELUNREGISTEREXECUTOR ////
//	extern sentinelExecutor *sentinelFindExecutor(const char *name, bool forDevice = true);
//	extern void sentinelRegisterExecutor(sentinelExecutor *exec, bool makeDefault = false, bool forDevice = true);
//	extern void sentinelUnregisterExecutor(sentinelExecutor *exec, bool forDevice = true);

//// SENTINELREGISTERFILEUTILS ////
//	extern void sentinelRegisterFileUtils();
